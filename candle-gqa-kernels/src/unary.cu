
#include <hip/hip_runtime.h>
#include<stdint.h>
#include "hip/hip_fp16.h"
#if __CUDA_ARCH__ >= 800
#include <cuda_bf16.h>
#endif

// Core repeat_kv function template for standard types
template<typename T>
__device__ void repeat_kv(
    const T* states, 
    T* repeated_states,
    const int n_local_heads,
    const int n_repeats,
    const int seqlen,
    const int head_dim
) {
    int seq_idx = blockIdx.x;
    int head_idx = blockIdx.y;
    int rep_idx = blockIdx.z;
    int dim_idx = threadIdx.x;

    int input_offset = head_idx * seqlen * head_dim + seq_idx * head_dim + dim_idx;
    int expanded_head_idx = head_idx * n_repeats + rep_idx;
    int output_offset = expanded_head_idx * seqlen * head_dim + seq_idx * head_dim + dim_idx;

    repeated_states[output_offset] = states[input_offset];
    // repeated_states[output_offset] = static_cast<T>(1);  // Set all elements to 1 as a test
}

    // const T* value_states,
    // const TYPENAME *value_states,    
    // T* repeated_values,
    // TYPENAME *repeated_values,       

// Macro to define repeat_kv kernel for each type
#define REPEAT_KV_OP(TYPENAME, FN_NAME) \
extern "C" __global__ void FN_NAME( \
    const TYPENAME *states,      \
    TYPENAME *repeated_states,       \
    const int n_local_heads,         \
    const int n_repeats,             \
    const int seqlen,                \
    const int head_dim) {            \
    repeat_kv(states, repeated_states, n_local_heads, n_repeats, seqlen, head_dim); \
}

REPEAT_KV_OP(float, repeat_kv_f32)
REPEAT_KV_OP(double, repeat_kv_f64)
REPEAT_KV_OP(uint8_t, repeat_kv_u8)
REPEAT_KV_OP(uint32_t, repeat_kv_u32)
REPEAT_KV_OP(int64_t, repeat_kv_i64)

#if __CUDA_ARCH__ >= 530
REPEAT_KV_OP(__half, repeat_kv_f16)
#endif

#if __CUDA_ARCH__ >= 800
REPEAT_KV_OP(__nv_bfloat16, repeat_kv_bf16)
#endif